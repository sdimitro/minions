#include <hip/hip_runtime.h>

#include <stdio.h>
#include <stdlib.h>
#include <time.h>
#include <math.h>
#include <thread>
#include <vector>
#include <chrono>
#include <algorithm>

// CUDA kernel for vector addition
__global__ void vectorAddGPU(float *a, float *b, float *c, int n) {
    int idx = blockIdx.x * blockDim.x + threadIdx.x;
    if (idx < n) {
        c[idx] = a[idx] + b[idx];
    }
}

// Single-threaded CPU version
void vectorAddCPU(float *a, float *b, float *c, int n) {
    for (int i = 0; i < n; i++) {
        c[i] = a[i] + b[i];
    }
}

// Multithreaded CPU version
void vectorAddCPUMultithreaded(float *a, float *b, float *c, int n, int numThreads) {
    std::vector<std::thread> threads;
    int elementsPerThread = n / numThreads;
    
    for (int t = 0; t < numThreads; t++) {
        int start = t * elementsPerThread;
        int end = (t == numThreads - 1) ? n : start + elementsPerThread;
        
        threads.emplace_back([=]() {
            for (int i = start; i < end; i++) {
                c[i] = a[i] + b[i];
            }
        });
    }
    
    for (auto& thread : threads) {
        thread.join();
    }
}

// Utility functions
void checkCudaError(hipError_t error, const char* message) {
    if (error != hipSuccess) {
        printf("CUDA Error: %s - %s\n", message, hipGetErrorString(error));
        exit(1);
    }
}

void initializeVector(float *vec, int n) {
    for (int i = 0; i < n; i++) {
        vec[i] = (float)rand() / RAND_MAX;
    }
}

double benchmarkCPU(void (*func)(float*, float*, float*, int), float *a, float *b, float *c, int n) {
    auto start = std::chrono::high_resolution_clock::now();
    func(a, b, c, n);
    auto end = std::chrono::high_resolution_clock::now();
    
    auto duration = std::chrono::duration_cast<std::chrono::microseconds>(end - start);
    return duration.count() / 1000000.0; // Convert to seconds
}

double benchmarkCPUMultithreaded(float *a, float *b, float *c, int n, int numThreads) {
    auto start = std::chrono::high_resolution_clock::now();
    vectorAddCPUMultithreaded(a, b, c, n, numThreads);
    auto end = std::chrono::high_resolution_clock::now();
    
    auto duration = std::chrono::duration_cast<std::chrono::microseconds>(end - start);
    return duration.count() / 1000000.0; // Convert to seconds
}

double benchmarkGPUWithMemory(float *h_a, float *h_b, float *h_c, int n, int blockSize) {
    const size_t bytes = n * sizeof(float);
    
    float *d_a, *d_b, *d_c;
    checkCudaError(hipMalloc(&d_a, bytes), "Allocating device memory for a");
    checkCudaError(hipMalloc(&d_b, bytes), "Allocating device memory for b");
    checkCudaError(hipMalloc(&d_c, bytes), "Allocating device memory for c");
    
    hipEvent_t start, stop;
    checkCudaError(hipEventCreate(&start), "Creating start event");
    checkCudaError(hipEventCreate(&stop), "Creating stop event");
    
    checkCudaError(hipEventRecord(start), "Recording start event");
    
    // Memory transfers + computation
    checkCudaError(hipMemcpy(d_a, h_a, bytes, hipMemcpyHostToDevice), "Copying a to device");
    checkCudaError(hipMemcpy(d_b, h_b, bytes, hipMemcpyHostToDevice), "Copying b to device");
    
    int gridSize = (n + blockSize - 1) / blockSize;
    vectorAddGPU<<<gridSize, blockSize>>>(d_a, d_b, d_c, n);
    
    checkCudaError(hipMemcpy(h_c, d_c, bytes, hipMemcpyDeviceToHost), "Copying result to host");
    
    checkCudaError(hipEventRecord(stop), "Recording stop event");
    checkCudaError(hipEventSynchronize(stop), "Synchronizing stop event");
    
    float milliseconds = 0;
    checkCudaError(hipEventElapsedTime(&milliseconds, start, stop), "Getting elapsed time");
    
    checkCudaError(hipFree(d_a), "Freeing device memory a");
    checkCudaError(hipFree(d_b), "Freeing device memory b");
    checkCudaError(hipFree(d_c), "Freeing device memory c");
    checkCudaError(hipEventDestroy(start), "Destroying start event");
    checkCudaError(hipEventDestroy(stop), "Destroying stop event");
    
    return milliseconds / 1000.0;
}

int main() {
    // Test different problem sizes
    std::vector<int> problemSizes = {1000, 10000, 100000, 1000000, 10000000, 100000000};
    
    printf("CPU vs GPU Vector Addition - The Reality Check\n");
    printf("=============================================\n\n");
    
    int numCores = std::thread::hardware_concurrency();
    printf("System info:\n");
    printf("  CPU cores available: %d\n", numCores);
    printf("  Testing problem sizes: 1K to 100M elements\n\n");
    
    for (int n : problemSizes) {
        size_t bytes = n * sizeof(float);
        printf("Problem size: %d elements (%.2f MB total)\n", n, (3.0 * bytes) / (1024.0 * 1024.0));
        printf("-----------------------------------------------\n");
        
        // Allocate memory
        float *h_a = (float*)malloc(bytes);
        float *h_b = (float*)malloc(bytes);
        float *h_c1 = (float*)malloc(bytes);
        float *h_c2 = (float*)malloc(bytes);
        float *h_c3 = (float*)malloc(bytes);
        
        if (!h_a || !h_b || !h_c1 || !h_c2 || !h_c3) {
            printf("Failed to allocate memory for size %d\n", n);
            continue;
        }
        
        // Initialize data
        srand(42); // Consistent seed for reproducible results
        initializeVector(h_a, n);
        initializeVector(h_b, n);
        
        // Benchmark single-threaded CPU
        double cpu_single_time = benchmarkCPU(vectorAddCPU, h_a, h_b, h_c1, n);
        
        // Benchmark multithreaded CPU
        double cpu_multi_time = benchmarkCPUMultithreaded(h_a, h_b, h_c2, n, numCores);
        
        // Benchmark GPU (including memory transfers)
        double gpu_time = benchmarkGPUWithMemory(h_a, h_b, h_c3, n, 256);
        
        // Calculate bandwidth
        double total_bytes = 3.0 * bytes; // Read A, Read B, Write C
        double cpu_single_bandwidth = total_bytes / (cpu_single_time * 1e9);
        double cpu_multi_bandwidth = total_bytes / (cpu_multi_time * 1e9);
        double gpu_bandwidth = total_bytes / (gpu_time * 1e9);
        
        printf("  Single-threaded CPU: %.6f s (%.2f GB/s)\n", cpu_single_time, cpu_single_bandwidth);
        printf("  Multi-threaded CPU:  %.6f s (%.2f GB/s) - %.2fx speedup\n", 
               cpu_multi_time, cpu_multi_bandwidth, cpu_single_time / cpu_multi_time);
        printf("  GPU (with transfers): %.6f s (%.2f GB/s) - %.2fx vs single CPU\n", 
               gpu_time, gpu_bandwidth, cpu_single_time / gpu_time);
        
        // Determine the winner
        if (cpu_multi_time < gpu_time) {
            printf("  🏆 WINNER: Multi-threaded CPU (%.2fx faster than GPU)\n", gpu_time / cpu_multi_time);
        } else {
            printf("  🏆 WINNER: GPU (%.2fx faster than multi-threaded CPU)\n", cpu_multi_time / gpu_time);
        }
        
        // Memory transfer overhead analysis
        if (n >= 100000) { // Only for larger sizes where GPU timing is more reliable
            // Estimate pure computation time (very rough)
            double estimated_transfer_time = gpu_time * 0.8; // Assume 80% is transfer
            double estimated_compute_time = gpu_time * 0.2;  // Assume 20% is compute
            printf("  GPU breakdown (estimated): %.2f%% memory transfer, %.2f%% computation\n", 
                   (estimated_transfer_time / gpu_time) * 100.0,
                   (estimated_compute_time / gpu_time) * 100.0);
        }
        
        printf("\n");
        
        // Cleanup
        free(h_a);
        free(h_b);
        free(h_c1);
        free(h_c2);
        free(h_c3);
    }
    
    printf("Key Takeaways:\n");
    printf("==============\n");
    printf("1. For small problems: CPU wins due to GPU memory transfer overhead\n");
    printf("2. For large problems: Multi-threaded CPU often still competitive\n");
    printf("3. Vector addition is memory-bound, not compute-bound\n");
    printf("4. GPU shines for compute-intensive problems with high arithmetic intensity\n");
    printf("5. Always compare against optimized CPU code, not single-threaded!\n\n");
    
    printf("Better GPU use cases:\n");
    printf("- Matrix multiplication\n");
    printf("- Image/signal processing\n");
    printf("- Machine learning training\n");
    printf("- Scientific simulations\n");
    printf("- Cryptography\n");
    printf("- Ray tracing\n");
    
    return 0;
}
