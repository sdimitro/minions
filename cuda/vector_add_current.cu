#include "hip/hip_runtime.h"
#include <hip/hip_runtime.h>
#include <>
#include <stdio.h>
#include <stdlib.h>
#include <time.h>
#include <math.h>

// CUDA kernel for vector addition
__global__ void vectorAdd(float *a, float *b, float *c, int n) {
    // Calculate global thread index
    int idx = blockIdx.x * blockDim.x + threadIdx.x;
    
    // Ensure we don't go out of bounds
    if (idx < n) {
        c[idx] = a[idx] + b[idx];
    }
}

// CPU version for comparison
void vectorAddCPU(float *a, float *b, float *c, int n) {
    for (int i = 0; i < n; i++) {
        c[i] = a[i] + b[i];
    }
}

// Utility function to check CUDA errors
void checkCudaError(hipError_t error, const char* message) {
    if (error != hipSuccess) {
        printf("CUDA Error: %s - %s\n", message, hipGetErrorString(error));
        exit(1);
    }
}

// Function to initialize vectors with random values
void initializeVector(float *vec, int n) {
    for (int i = 0; i < n; i++) {
        vec[i] = (float)rand() / RAND_MAX;
    }
}

// Function to verify results
bool verifyResults(float *cpu_result, float *gpu_result, int n) {
    const float epsilon = 1e-5;
    for (int i = 0; i < n; i++) {
        if (fabs(cpu_result[i] - gpu_result[i]) > epsilon) {
            printf("Verification failed at index %d: CPU=%.6f, GPU=%.6f\n", 
                   i, cpu_result[i], gpu_result[i]);
            return false;
        }
    }
    return true;
}

// Benchmark function (kernel only)
double benchmarkGPU(float *d_a, float *d_b, float *d_c, int n, int blockSize) {
    int gridSize = (n + blockSize - 1) / blockSize;
    
    // Create CUDA events for timing
    hipEvent_t start, stop;
    checkCudaError(hipEventCreate(&start), "Creating start event");
    checkCudaError(hipEventCreate(&stop), "Creating stop event");
    
    // Warm-up run
    vectorAdd<<<gridSize, blockSize>>>(d_a, d_b, d_c, n);
    checkCudaError(hipDeviceSynchronize(), "Warmup synchronization");
    
    // Benchmark run
    checkCudaError(hipEventRecord(start), "Recording start event");
    vectorAdd<<<gridSize, blockSize>>>(d_a, d_b, d_c, n);
    checkCudaError(hipEventRecord(stop), "Recording stop event");
    checkCudaError(hipEventSynchronize(stop), "Synchronizing stop event");
    
    float milliseconds = 0;
    checkCudaError(hipEventElapsedTime(&milliseconds, start, stop), "Getting elapsed time");
    
    // Cleanup events
    checkCudaError(hipEventDestroy(start), "Destroying start event");
    checkCudaError(hipEventDestroy(stop), "Destroying stop event");
    
    return milliseconds / 1000.0; // Convert to seconds
}

// Benchmark function including memory transfers
double benchmarkGPUWithMemory(float *h_a, float *h_b, float *h_c, int n, int blockSize) {
    const size_t bytes = n * sizeof(float);
    int gridSize = (n + blockSize - 1) / blockSize;
    
    // Create CUDA events for timing
    hipEvent_t start, stop;
    checkCudaError(hipEventCreate(&start), "Creating start event");
    checkCudaError(hipEventCreate(&stop), "Creating stop event");
    
    // Allocate device memory
    float *d_a, *d_b, *d_c;
    checkCudaError(hipMalloc(&d_a, bytes), "Allocating device memory for a");
    checkCudaError(hipMalloc(&d_b, bytes), "Allocating device memory for b");
    checkCudaError(hipMalloc(&d_c, bytes), "Allocating device memory for c");
    
    // Time the entire GPU pipeline
    checkCudaError(hipEventRecord(start), "Recording start event");
    
    // Copy data to device
    checkCudaError(hipMemcpy(d_a, h_a, bytes, hipMemcpyHostToDevice), "Copying a to device");
    checkCudaError(hipMemcpy(d_b, h_b, bytes, hipMemcpyHostToDevice), "Copying b to device");
    
    // Launch kernel
    vectorAdd<<<gridSize, blockSize>>>(d_a, d_b, d_c, n);
    
    // Copy result back
    checkCudaError(hipMemcpy(h_c, d_c, bytes, hipMemcpyDeviceToHost), "Copying result to host");
    
    checkCudaError(hipEventRecord(stop), "Recording stop event");
    checkCudaError(hipEventSynchronize(stop), "Synchronizing stop event");
    
    float milliseconds = 0;
    checkCudaError(hipEventElapsedTime(&milliseconds, start, stop), "Getting elapsed time");
    
    // Cleanup
    checkCudaError(hipFree(d_a), "Freeing device memory a");
    checkCudaError(hipFree(d_b), "Freeing device memory b");
    checkCudaError(hipFree(d_c), "Freeing device memory c");
    checkCudaError(hipEventDestroy(start), "Destroying start event");
    checkCudaError(hipEventDestroy(stop), "Destroying stop event");
    
    return milliseconds / 1000.0; // Convert to seconds
}

int main() {
    // Vector size
    const int n = 1000000000; 
    const size_t bytes = n * sizeof(float);
    
    printf("CUDA Vector Addition Tutorial\n");
    printf("Vector size: %d elements (%.2f MB per vector)\n", n, bytes / (1024.0 * 1024.0));
    printf("===============================================\n\n");
    
    // Allocate host memory
    float *h_a = (float*)malloc(bytes);
    float *h_b = (float*)malloc(bytes);
    float *h_c_cpu = (float*)malloc(bytes);
    float *h_c_gpu = (float*)malloc(bytes);
    
    if (!h_a || !h_b || !h_c_cpu || !h_c_gpu) {
        printf("Failed to allocate host memory\n");
        return 1;
    }
    
    // Initialize vectors
    srand(time(NULL));
    initializeVector(h_a, n);
    initializeVector(h_b, n);
    
    // CPU benchmark
    printf("1. CPU Benchmark\n");
    clock_t cpu_start = clock();
    vectorAddCPU(h_a, h_b, h_c_cpu, n);
    clock_t cpu_end = clock();
    double cpu_time = ((double)(cpu_end - cpu_start)) / CLOCKS_PER_SEC;
    printf("   CPU time: %.6f seconds\n\n", cpu_time);
    
    // GPU setup
    printf("2. GPU Setup and Memory Management\n");
    
    // Allocate device memory
    float *d_a, *d_b, *d_c;
    checkCudaError(hipMalloc(&d_a, bytes), "Allocating device memory for a");
    checkCudaError(hipMalloc(&d_b, bytes), "Allocating device memory for b");
    checkCudaError(hipMalloc(&d_c, bytes), "Allocating device memory for c");
    printf("   ✓ Allocated %.2f MB on GPU\n", 3 * bytes / (1024.0 * 1024.0));
    
    // Copy data to device
    checkCudaError(hipMemcpy(d_a, h_a, bytes, hipMemcpyHostToDevice), "Copying a to device");
    checkCudaError(hipMemcpy(d_b, h_b, bytes, hipMemcpyHostToDevice), "Copying b to device");
    printf("   ✓ Copied input data to GPU\n\n");
    
    // Test different block sizes
    printf("3. GPU Benchmarks - Kernel Only\n");
    int blockSizes[] = {32, 64, 128, 256, 512, 1024};
    int numBlockSizes = sizeof(blockSizes) / sizeof(blockSizes[0]);
    
    double bestTime = 1e9;
    int bestBlockSize = 0;
    
    for (int i = 0; i < numBlockSizes; i++) {
        int blockSize = blockSizes[i];
        int gridSize = (n + blockSize - 1) / blockSize;
        
        double gpu_time = benchmarkGPU(d_a, d_b, d_c, n, blockSize);
        
        printf("   Block size: %4d, Grid size: %6d, Time: %.6f s, Speedup: %.2fx\n", 
               blockSize, gridSize, gpu_time, cpu_time / gpu_time);
        
        if (gpu_time < bestTime) {
            bestTime = gpu_time;
            bestBlockSize = blockSize;
        }
    }
    
    printf("\n   Best kernel configuration: Block size %d (%.6f seconds, %.2fx speedup)\n\n", 
           bestBlockSize, bestTime, cpu_time / bestTime);
    
    // Test with memory transfers included
    printf("4. GPU Benchmarks - Including Memory Transfers\n");
    for (int i = 0; i < numBlockSizes; i++) {
        int blockSize = blockSizes[i];
        
        double gpu_time_with_mem = benchmarkGPUWithMemory(h_a, h_b, h_c_gpu, n, blockSize);
        
        printf("   Block size: %4d, Total time: %.6f s, Speedup: %.2fx\n", 
               blockSize, gpu_time_with_mem, cpu_time / gpu_time_with_mem);
    }
    printf("\n");
    
    // Run with best configuration and verify results
    printf("5. Result Verification\n");
    int bestGridSize = (n + bestBlockSize - 1) / bestBlockSize;
    vectorAdd<<<bestGridSize, bestBlockSize>>>(d_a, d_b, d_c, n);
    checkCudaError(hipDeviceSynchronize(), "Final kernel execution");
    
    // Copy result back to host
    checkCudaError(hipMemcpy(h_c_gpu, d_c, bytes, hipMemcpyDeviceToHost), "Copying result to host");
    
    // Verify results
    if (verifyResults(h_c_cpu, h_c_gpu, n)) {
        printf("   ✓ Results verified successfully!\n\n");
    } else {
        printf("   ✗ Result verification failed!\n\n");
    }
    
    // Performance analysis
    printf("6. Performance Analysis\n");
    double bandwidth_cpu = (3.0 * bytes) / (cpu_time * 1e9); // GB/s
    double bandwidth_gpu = (3.0 * bytes) / (bestTime * 1e9); // GB/s
    printf("   CPU effective bandwidth: %.2f GB/s\n", bandwidth_cpu);
    printf("   GPU effective bandwidth: %.2f GB/s\n", bandwidth_gpu);
    printf("   Bandwidth improvement: %.2fx\n\n", bandwidth_gpu / bandwidth_cpu);
    
    // GPU device information
    printf("7. GPU Device Information\n");
    hipDeviceProp_t prop;
    checkCudaError(hipGetDeviceProperties(&prop, 0), "Getting device properties");
    printf("   Device: %s\n", prop.name);
    printf("   Compute capability: %d.%d\n", prop.major, prop.minor);
    printf("   Max threads per block: %d\n", prop.maxThreadsPerBlock);
    printf("   Max grid size: %d x %d x %d\n", prop.maxGridSize[0], prop.maxGridSize[1], prop.maxGridSize[2]);
    printf("   Memory bandwidth: %.2f GB/s\n", 2.0 * prop.memoryClockRate * (prop.memoryBusWidth / 8) / 1.0e6);
    
    // Cleanup
    free(h_a);
    free(h_b);
    free(h_c_cpu);
    free(h_c_gpu);
    
    checkCudaError(hipFree(d_a), "Freeing device memory a");
    checkCudaError(hipFree(d_b), "Freeing device memory b");
    checkCudaError(hipFree(d_c), "Freeing device memory c");
    
    printf("\nTutorial completed successfully!\n");
    return 0;
}
