#include <hip/hip_runtime.h>

#include <stdio.h>
#include <stdlib.h>
#include <time.h>
#include <math.h>

// Compute-intensive kernel: Matrix-vector multiplication with expensive operations
__global__ void computeIntensiveKernel(float *input, float *output, int n) {
    int idx = blockIdx.x * blockDim.x + threadIdx.x;
    
    if (idx < n) {
        float result = input[idx];
        
        // Perform many expensive operations to make it compute-bound
        for (int i = 0; i < 1000; i++) {
            result = sinf(result) * cosf(result) + sqrtf(fabsf(result));
            result = powf(result, 0.9f) + logf(fabsf(result) + 1.0f);
            result = expf(result * 0.001f) - tanhf(result * 0.01f);
        }
        
        output[idx] = result;
    }
}

// CPU version of the same computation
void computeIntensiveCPU(float *input, float *output, int n) {
    for (int i = 0; i < n; i++) {
        float result = input[i];
        
        for (int j = 0; j < 1000; j++) {
            result = sinf(result) * cosf(result) + sqrtf(fabsf(result));
            result = powf(result, 0.9f) + logf(fabsf(result) + 1.0f);
            result = expf(result * 0.001f) - tanhf(result * 0.01f);
        }
        
        output[i] = result;
    }
}

// Simple kernel that shows warp divergence effects
__global__ void warpDivergenceKernel(float *input, float *output, int n) {
    int idx = blockIdx.x * blockDim.x + threadIdx.x;
    
    if (idx < n) {
        float result = input[idx];
        
        // This creates warp divergence - different threads take different paths
        // Each path now performs the same amount of work as computeIntensiveKernel
        // A warp will execute both paths sequentially, highlighting the overhead.
        if (threadIdx.x % 2 == 0) {
            // Even threads do one type of computation
            for (int i = 0; i < 1000; i++) { // Increased iterations and operations
                result = sinf(result) * cosf(result) + sqrtf(fabsf(result));
                result = powf(result, 0.9f) + logf(fabsf(result) + 1.0f);
                result = expf(result * 0.001f) - tanhf(result * 0.01f);
            }
        } else {
            // Odd threads do different computation
            for (int i = 0; i < 1000; i++) { // Increased iterations and operations
                result = sinf(result) * cosf(result) + sqrtf(fabsf(result));
                result = powf(result, 0.9f) + logf(fabsf(result) + 1.0f);
                result = expf(result * 0.001f) - tanhf(result * 0.01f);
            }
        }
        
        output[idx] = result;
    }
}

// Dedicated kernel for simple memory-bound copy operation
__global__ void copyKernel(float *input, float *output, int n) {
    int idx = blockIdx.x * blockDim.x + threadIdx.x;
    if (idx < n) {
        output[idx] = input[idx];
    }
}


// Utility functions
void checkCudaError(hipError_t error, const char* message) {
    if (error != hipSuccess) {
        printf("CUDA Error: %s - %s\n", message, hipGetErrorString(error));
        exit(1);
    }
}

void initializeVector(float *vec, int n) {
    for (int i = 0; i < n; i++) {
        vec[i] = 0.1f + (float)i * 0.001f; // Avoid problematic values for math functions
    }
}

// Modified benchmarkGPU to accept a function pointer to a __global__ kernel
// This function benchmarks ONLY the kernel execution time.
double benchmarkGPU(void (*kernel)(float*, float*, int), float *d_in, float *d_out, int n, int blockSize) {
    int gridSize = (n + blockSize - 1) / blockSize;
    
    hipEvent_t start, stop;
    checkCudaError(hipEventCreate(&start), "Creating start event");
    checkCudaError(hipEventCreate(&stop), "Creating stop event");
    
    // Warm-up
    kernel<<<gridSize, blockSize>>>(d_in, d_out, n);
    checkCudaError(hipDeviceSynchronize(), "Warmup synchronization");
    
    // Benchmark
    checkCudaError(hipEventRecord(start), "Recording start event");
    kernel<<<gridSize, blockSize>>>(d_in, d_out, n);
    checkCudaError(hipEventRecord(stop), "Recording stop event");
    checkCudaError(hipEventSynchronize(stop), "Synchronizing stop event");
    
    float milliseconds = 0;
    checkCudaError(hipEventElapsedTime(&milliseconds, start, stop), "Getting elapsed time");
    
    checkCudaError(hipEventDestroy(start), "Destroying start event");
    checkCudaError(hipEventDestroy(stop), "Destroying stop event");
    
    return milliseconds / 1000.0; // Return time in seconds
}

int main() {
    const int n = 100000; // Smaller size for compute-intensive operations
    const size_t bytes = n * sizeof(float);
    
    printf("CUDA Warp Efficiency Demonstration\n");
    printf("Problem size: %d elements\n", n);
    printf("===============================================\n\n");
    
    // Allocate host memory
    float *h_input = (float*)malloc(bytes);
    float *h_output_cpu = (float*)malloc(bytes);
    float *h_output_gpu = (float*)malloc(bytes);
    
    initializeVector(h_input, n);
    
    // CPU benchmark
    printf("1. CPU Compute-Intensive Benchmark\n");
    clock_t cpu_start = clock();
    computeIntensiveCPU(h_input, h_output_cpu, n);
    clock_t cpu_end = clock();
    double cpu_time = ((double)(cpu_end - cpu_start)) / CLOCKS_PER_SEC;
    printf("    CPU time: %.4f seconds\n\n", cpu_time);
    
    // GPU setup
    float *d_input, *d_output;
    checkCudaError(hipMalloc(&d_input, bytes), "Allocating device input");
    checkCudaError(hipMalloc(&d_output, bytes), "Allocating device output");
    
    // Events for timing memory transfers
    hipEvent_t start_h2d, stop_h2d;
    hipEvent_t start_d2h, stop_d2h;
    checkCudaError(hipEventCreate(&start_h2d), "Creating H2D start event");
    checkCudaError(hipEventCreate(&stop_h2d), "Creating H2D stop event");
    checkCudaError(hipEventCreate(&start_d2h), "Creating D2H start event");
    checkCudaError(hipEventCreate(&stop_d2h), "Creating D2H stop event");

    // Copy input to device and time it (H2D)
    checkCudaError(hipEventRecord(start_h2d), "Recording H2D start event");
    checkCudaError(hipMemcpy(d_input, h_input, bytes, hipMemcpyHostToDevice), "Copying input to device");
    checkCudaError(hipEventRecord(stop_h2d), "Recording H2D stop event");
    checkCudaError(hipEventSynchronize(stop_h2d), "Synchronizing H2D stop event");
    float h2d_milliseconds = 0;
    checkCudaError(hipEventElapsedTime(&h2d_milliseconds, start_h2d, stop_h2d), "Getting H2D elapsed time");
    double h2d_time = h2d_milliseconds / 1000.0; // Declare h2d_time here, accessible globally in main
    
    // Variables to store kernel and D2H times for the specific 256 block size run
    double regular_kernel_time_256 = 0;
    double d2h_time_256 = 0;

    // Test compute-intensive kernel with different block sizes
    printf("2. GPU Compute-Intensive Kernel - Block Size Effects (including memory transfers)\n");
    int blockSizes[] = {31, 32, 63, 64, 127, 128, 255, 256, 511, 512};
    int numBlockSizes = sizeof(blockSizes) / sizeof(blockSizes[0]);
    
    for (int i = 0; i < numBlockSizes; i++) {
        int blockSize = blockSizes[i];
        
        // Benchmark kernel execution time only
        double kernel_exec_time = benchmarkGPU(computeIntensiveKernel, d_input, d_output, n, blockSize);
        
        // Copy output back to host and time it (D2H)
        checkCudaError(hipEventRecord(start_d2h), "Recording D2H start event");
        checkCudaError(hipMemcpy(h_output_gpu, d_output, bytes, hipMemcpyDeviceToHost), "Copying output to host");
        checkCudaError(hipEventRecord(stop_d2h), "Recording D2H stop event");
        checkCudaError(hipEventSynchronize(stop_d2h), "Synchronizing D2H stop event");
        float d2h_milliseconds = 0;
        checkCudaError(hipEventElapsedTime(&d2h_milliseconds, start_d2h, stop_d2h), "Getting D2H elapsed time");
        double d2h_time = d2h_milliseconds / 1000.0; // This d2h_time is local to the loop

        // Store the times for blockSize 256 for later use
        if (blockSize == 256) {
            regular_kernel_time_256 = kernel_exec_time;
            d2h_time_256 = d2h_time;
        }

        // Total GPU time including transfers
        double total_gpu_time = h2d_time + kernel_exec_time + d2h_time;
        
        bool isMultipleOf32 = (blockSize % 32 == 0);
        int wastedThreads = isMultipleOf32 ? 0 : (32 - (blockSize % 32));
        
        printf("    Block size: %3d, Kernel Time: %.4f s, H2D Time: %.4f s, D2H Time: %.4f s, Total GPU Time: %.4f s, Speedup: %5.1fx %s\n", 
               blockSize, kernel_exec_time, h2d_time, d2h_time, total_gpu_time, cpu_time / total_gpu_time,
               isMultipleOf32 ? "✓" : "✗");
        
        if (!isMultipleOf32) {
            printf("        └─ Wasted threads per partial warp: %d/32 (%.1f%% efficiency)\n", 
                   wastedThreads, ((32.0 - wastedThreads) / 32.0) * 100.0);
        }
    }
    
    printf("\n3. Warp Divergence Effects\n");
    printf("    Testing kernel with branching (warp divergence)...\n");
    
    // Compare regular vs divergent kernels (kernel execution time only)
    double regular_kernel_time = benchmarkGPU(computeIntensiveKernel, d_input, d_output, n, 256);
    double divergent_kernel_time = benchmarkGPU(warpDivergenceKernel, d_input, d_output, n, 256);
    
    printf("    Regular kernel (256 threads):    %.4f s\n", regular_kernel_time);
    printf("    Divergent kernel (256 threads):  %.4f s\n", divergent_kernel_time);
    printf("    Divergence overhead: %.1f%%\n\n", ((divergent_kernel_time - regular_kernel_time) / regular_kernel_time) * 100.0);
    
    // Memory bandwidth analysis
    printf("4. Memory Bandwidth Analysis\n");
    
    printf("    Testing memory-bound operation (simple copy)...\n");
    
    // Now calling the standalone __global__ copyKernel
    double copy_kernel_time = benchmarkGPU(copyKernel, d_input, d_output, n, 256);
    
    // Calculate effective bandwidth
    double bytes_transferred = 2.0 * bytes; // Read input, write output
    double bandwidth = bytes_transferred / (copy_kernel_time * 1e9); // GB/s
    
    printf("    Copy kernel time: %.6f s\n", copy_kernel_time);
    printf("    Effective bandwidth: %.2f GB/s\n", bandwidth);
    printf("    Compute-to-memory ratio: %.1fx\n\n", regular_kernel_time / copy_kernel_time);
    
    printf("5. Why Vector Addition Isn't Great for GPU\n");
    printf("    Vector addition characteristics:\n");
    printf("    - Memory bandwidth bound (not compute bound)\n");
    printf("    - Low arithmetic intensity (1 operation per 2 memory reads + 1 write)\n");
    printf("    - Memory transfer overhead dominates for small/medium arrays\n");
    printf("    - CPU cache hierarchy often more efficient for this pattern\n\n");
    
    printf("6. When GPU Acceleration Makes Sense\n");
    printf("    ✓ High arithmetic intensity (many operations per memory access)\n");
    printf("    ✓ Large datasets (amortize memory transfer costs)\n");
    printf("    ✓ Parallelizable algorithms (no dependencies between threads)\n");
    printf("    ✓ Regular memory access patterns (coalesced memory access)\n");
    printf("    ✓ Compute-bound operations (like our intensive kernel: %.1fx speedup, including transfers)\n\n", 
             cpu_time / (regular_kernel_time_256 + h2d_time + d2h_time_256)); // Use stored times for final speedup
    
    // Cleanup
    free(h_input);
    free(h_output_cpu);
    free(h_output_gpu);
    checkCudaError(hipFree(d_input), "Freeing device input");
    checkCudaError(hipFree(d_output), "Freeing device output");
    checkCudaError(hipEventDestroy(start_h2d), "Destroying H2D start event");
    checkCudaError(hipEventDestroy(stop_h2d), "Destroying H2D stop event");
    checkCudaError(hipEventDestroy(start_d2h), "Destroying D2H start event");
    checkCudaError(hipEventDestroy(stop_d2h), "Destroying D2H stop event");
    
    return 0;
}

